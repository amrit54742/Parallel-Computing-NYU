#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <time.h>

// Parallel Computing Lab 3
// Author: Andrew Huang

// forward declare
void deviceProperties(void);
long getMax(long * a, long);

#define THREADS_PER_BLOCK 1024 // 3.x

void deviceProperties(void){ // displays device properties
    int nDevices;
    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (KHz): %d\n",
            prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n",
            prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
            2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    }
}
__global__ void getMaxCUDA(long arr[], long size, long result[]){ // Cuda CALL kernal func
    __shared__ long arr_all[THREADS_PER_BLOCK];
    long gid = blockIdx.x * blockDim.x + threadIdx.x;
    arr_all[threadIdx.x] = -INT_MAX;
    if (gid < size){
        arr_all[threadIdx.x] = arr[gid]; // bounds
    }
    __syncthreads();
    for (long s = blockIdx.x/2; s >0; s = s/2){
        __syncthreads();
        if (threadIdx.x < s && gid < size) {
            arr_all[threadIdx.x] = max(arr_all[threadIdx.x], arr_all[threadIdx.x + s]);
        }
    }
    if (threadIdx.x == 0)result[blockIdx.x] = arr_all[0];
}

long getMax(long arr[], long size){ // array and n;
   // safetogo
    long * new_arr; // this is due to overflow
    long * answer;
    long * result;
    long * arr_copy; // we have to make a copy to the device
    long new_size;
    if (size % THREADS_PER_BLOCK != 0) {
        new_size = (size / THREADS_PER_BLOCK + 1) * THREADS_PER_BLOCK;
    } else {
        new_size = size;
    }
    new_arr = (long *) malloc(sizeof(long) * new_size);
    for (long i = 0; i < new_size;i++){
        if (i < size){
            new_arr[i] = arr[i];
        } else {
            new_arr[i]=0;
        }
    }
    long block_count = new_size / THREADS_PER_BLOCK;
    hipMalloc((void **) &arr_copy, sizeof(long) *new_size);
    hipMemcpy((void *) arr_copy, (void *) new_arr, sizeof(long)*new_size, hipMemcpyHostToDevice);
    hipMalloc((void **) &result, sizeof(long) *block_count); // block results
    do {
        block_count = ceil((float)new_size / (float)THREADS_PER_BLOCK);
        getMaxCUDA<<<block_count, THREADS_PER_BLOCK>>>(arr_copy, new_size, result);
        new_size = block_count;
        arr_copy = result;
     } while (block_count > 1);
    answer = (long*) malloc(sizeof(long) * block_count);
    hipMemcpy((void *)answer, (void *)result, block_count * sizeof(long),hipMemcpyDeviceToHost);
    long res =answer[0];
    hipFree(result);
    hipFree(arr_copy);
    free(new_arr);
    free(answer);
    return res;
}

int main(int argc, char * argv[]){
    if (argc!= 2){
        printf("Usage: maxgpu N\n");
        printf("where N is the size of the array");
        exit(1);
    }
    int n; // number of integers and size
    long *arr; // array
    n = atoi(argv[1]);

    arr = (long *)malloc(sizeof(long) * n);
    if (!arr){
        printf("failed to allocate array\n");
        exit(1);
    }
    srand(time(NULL));
    for (long i = 0; i < n; i ++){
        arr[i] = rand() % n; 
    }
    // cuda time
    //deviceProperties();
    long res = getMax(arr, n);
    printf("The maximum number in the array is: %ld\n", res);
    free(arr);
    return 0;
} 
